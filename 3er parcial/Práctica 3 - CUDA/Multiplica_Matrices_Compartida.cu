
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Multiplica_Matrices_SM(float *C, float *A, float *B, int nfil, int ncol)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int BLOCK_SIZE = 4;

	int aBegin = ncol * BLOCK_SIZE * by;
	int aEnd = aBegin + ncol - 1;
	int aStep = BLOCK_SIZE;

	int bBegin = BLOCK_SIZE * bx;
	int bStep = BLOCK_SIZE * ncol;

	float sum_sub = 0.0f;

	for(int a = aBegin, b = bBegin ; a <= aEnd ; a+= aStep, b += bStep)
	{
		__shared__ float As[4][4];
		__shared__ float Bs[4][4];

		As[ty][tx] = A[a + ncol * ty + tx];
		Bs[ty][tx] = B[b + ncol * ty + tx];

		__syncthreads();

		for(int k = 0 ; k < BLOCK_SIZE ; k++)
		{
			sum_sub += As[ty][k] * Bs[k][tx];
		}

		__syncthreads();
	}

	int c = ncol * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + ncol * ty + tx] = sum_sub;
}

int div_up(int a, int b)
{
    if (a % b)  /* does a divide b leaving a remainder? */
        return a / b + 1; /* add in additional block */
    else
        return a / b; /* divides cleanly */
}

int main(void)
{
	float *A_h,*B_h,*C_h;
	float *A_d,*B_d,*C_d;
	int nfil = 12;
	int ncol = 12;
	int BLOCK_SIZE = 4;
	int N=nfil*ncol;

	size_t size=N * sizeof(float);

	A_h = (float *)malloc(size);
	B_h = (float *)malloc(size);
	C_h = (float *)malloc(size);

	for(int i=0; i<nfil; i++)
	{
		for(int j=0;j<ncol;j++)
		{
			A_h[i*ncol+j] = 1.0f;
			B_h[i*ncol+j] = 2.0f;
		}
	}

	hipMalloc((void **) &A_d, size);
	hipMalloc((void **) &B_d, size);
	hipMalloc((void **) &C_d, size);

	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

	dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
	dim3 n_blocks(div_up(ncol,block_size.x),div_up(nfil,block_size.y));

	Multiplica_Matrices_SM<<< n_blocks, block_size >>> (C_d,A_d,B_d,nfil,ncol);

	hipMemcpy(C_h,C_d,size,hipMemcpyDeviceToHost);

	printf("\n\nMatriz c:\n");

	for(int i=0; i<10; i++)
	{
		for(int j=0; j<10; j++)
		{
			printf("%.2f ", C_h[i*ncol+j]);
		}
		printf("\n");
	}

	free(A_h);
	free(B_h);
	free(C_h);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return(0);
}