
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Multiplca_Matrices_GM(float *C,float *A,float *B,int nfil,int ncol)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int index = idy*ncol+idx;

	if(idy<nfil && idx<ncol)
	{
		float sum=0.0f;

		for(int k=0;k<ncol;k++)
		{
			sum+=A[idy*ncol+k]*B[k*ncol+idx];
		}
		C[index] = sum;
	}
}

int div_up(int a, int b)
{
    if (a % b)  /* does a divide b leaving a remainder? */
        return a / b + 1; /* add in additional block */
    else
        return a / b; /* divides cleanly */
}

int main(void)
{
	float *A_h,*B_h,*C_h;
	float *A_d,*B_d,*C_d;
	int nfil = 12;
	int ncol = 12;
	int BLOCK_SIZE = 4;
	int N=nfil*ncol;

	size_t size=N * sizeof(float);

	A_h = (float *)malloc(size);
	B_h = (float *)malloc(size);
	C_h = (float *)malloc(size);

	for(int i=0; i<nfil; i++)
	{
		for(int j=0;j<ncol;j++)
		{
			A_h[i*ncol+j] = 1.0f;
			B_h[i*ncol+j] = 2.0f;
		}
	}

	hipMalloc((void **) &A_d, size);
	hipMalloc((void **) &B_d, size);
	hipMalloc((void **) &C_d, size);

	hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

	dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
	dim3 n_blocks(div_up(ncol,block_size.x),div_up(nfil,block_size.y));

	Multiplca_Matrices_GM<<< n_blocks, block_size >>> (C_d,A_d,B_d,nfil,ncol);

	hipMemcpy(C_h,C_d,size,hipMemcpyDeviceToHost);

	printf("\n\nMatriz c:\n");

	for(int i=0; i<10; i++)
	{
		for(int j=0; j<10; j++)
		{
			printf("%.2f ", C_h[i*ncol+j]);
		}
		printf("\n");
	}

	free(A_h);
	free(B_h);
	free(C_h);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return(0);
}