
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA(float e)
{
	printf("Hello, I am thread %d of block %d with value e = %f\n", threadIdx.x, blockIdx.x, e);
}

int main(int argc, char **argv)
{
	helloCUDA<<<3, 4>>>(2.71828f);

	hipDeviceReset();
	system("pause");
	return(0);
}